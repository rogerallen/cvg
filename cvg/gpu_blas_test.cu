#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <windows.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cublasXt.h>
#include "gpu_blas_test.h"
#include "util.h"

#define HANDLE_CUDA_ERROR( err ) ( HandleCudaError( err, __FILE__, __LINE__ ) )
static void HandleCudaError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_CUBLAS_ERROR( err, str ) ( HandleCublasError( err, __FILE__, __LINE__, str) )
static void HandleCublasError(hipblasStatus_t err, const char *file, int line, const char *str)
{
    if (err != HIPBLAS_STATUS_SUCCESS)
    {
        printf("error %s %d in %s at line %d\n", str, err, // why no cublasGetErrorString?
            file, line);
        exit(EXIT_FAILURE);
    }
}

void list_cuda_devices() 
{
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
    }
}

int cublas_gpu_test(int loops, int M, int N, int K)
{
    printf("NVIDIA CUBLAS sgemm: loops=%d M=%d N=%d K=%d\n", loops, M, N, K);
    
    list_cuda_devices();

    hipblasHandle_t handle;
    HANDLE_CUBLAS_ERROR(hipblasCreate(&handle),"hipblasCreate fail");

    float *a, *b, *c;
    new_float_matrix(a, M, K);
    new_float_matrix(b, K, N);
    new_float_matrix(c, M, N);

    // time all the extra stuff for setting up the matrices
    clock_t start, stop;
    start = clock();
    float *dev_a, *dev_b, *dev_c;
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_a, M*K*sizeof(*a)));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_b, K*N*sizeof(*b)));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_c, M*N*sizeof(*c)));
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(M, K, sizeof(*a), a, M, dev_a, M), "hipblasSetMatrix A fail");
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(K, N, sizeof(*b), b, K, dev_b, K), "hipblasSetMatrix B fail");
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(M, N, sizeof(*c), c, M, dev_c, M), "hipblasSetMatrix C fail");

    float alpha = 1.11f, beta = 0.91f;
    for (int i = 0; i < loops; ++i) {
        HANDLE_CUBLAS_ERROR(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, dev_a, M, dev_b, K, &beta, dev_c, M), "Sgemm fail");
    }
    HANDLE_CUBLAS_ERROR(hipblasGetMatrix(M, N, sizeof(*c), dev_c, M, c, M), "hipblasGetMatrix C fail");
    stop = clock();

    summarize(c, loops, M, N, K, start, stop);

    delete_float_matrix(a);
    delete_float_matrix(b);
    delete_float_matrix(c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipblasDestroy(handle);

    return 0;

}

int cublasxt_gpu_test(int loops, int M, int N, int K, int block_dim)
{
    printf("NVIDIA CUBLASXT sgemm: loops=%d M=%d N=%d K=%d block_dim=%d\n", loops, M, N, K, block_dim);

    list_cuda_devices();

    cublasXtHandle_t handle;
    HANDLE_CUBLAS_ERROR(cublasXtCreate(&handle), "cublasXtCreate fail");
    
    // NOTE: adjust this for your particular GPU configuration.
    int devices[1] = { 0 };
    HANDLE_CUBLAS_ERROR(cublasXtDeviceSelect(handle, 1, devices), "cublasXtDeviceSelect fail");

    HANDLE_CUBLAS_ERROR(cublasXtSetBlockDim(handle, block_dim), "cublasXtSetBlockDim fail");
    
    float *a, *b, *c;
    new_float_matrix(a, M, K);
    new_float_matrix(b, K, N);
    new_float_matrix(c, M, N);

    clock_t start, stop;
    start = clock();
    float alpha = 1.11f, beta = 0.91f;
    for (int i = 0; i < loops; ++i) {
        HANDLE_CUBLAS_ERROR(cublasXtSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, a, M, b, K, &beta, c, M), "Sgemm fail");
    }
    stop = clock();

    summarize(c, loops, M, N, K, start, stop);

    delete_float_matrix(a);
    delete_float_matrix(b);
    delete_float_matrix(c);

    cublasXtDestroy(handle);

    return 0;

}