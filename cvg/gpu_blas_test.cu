#include "hip/hip_runtime.h"
#include <assert.h>
#include <time.h>
#include <stdio.h>
// FIXME confirm this is needed
#ifndef NO_WINDOWS
#include <windows.h>
#endif
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cublasXt.h>
#include "gpu_blas_test.h"
#include "util.h"

#define HANDLE_CUDA_ERROR( err ) ( HandleCudaError( err, __FILE__, __LINE__ ) )
static void HandleCudaError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_CUBLAS_ERROR( err, str ) ( HandleCublasError( err, __FILE__, __LINE__, str) )
static void HandleCublasError(hipblasStatus_t err, const char *file, int line, const char *str)
{
    if (err != HIPBLAS_STATUS_SUCCESS)
    {
        printf("error %s %d in %s at line %d\n", str, err, // why no cublasGetErrorString?
            file, line);
        exit(EXIT_FAILURE);
    }
}

void list_cuda_devices()
{
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
    }
}

int gpu_cublas_sgemm(int loops, int M, int N, int K, float alpha, float beta, bool csv_output)
{
    if(!csv_output) {
        printf("NVIDIA CUBLAS sgemm: loops=%d M=%d N=%d K=%d alpha=%f beta=%f\n", loops, M, N, K, alpha, beta);

        list_cuda_devices();
    } else {
        printf("NVIDIA CUBLAS sgemm,%d,%d,%d,%d,%f,%f",loops, M, N, K, alpha, beta);
    }

    hipblasHandle_t handle;
    HANDLE_CUBLAS_ERROR(hipblasCreate(&handle),"hipblasCreate fail");

    float *a, *b, *c;
    new_float_matrix(a, M, K);
    new_float_matrix(b, K, N);
    new_float_matrix(c, M, N);

    // time all the extra stuff for setting up the matrices
    clock_t start, stop;
    clock_t start2, stop2;
    start = clock();
    float *dev_a, *dev_b, *dev_c;
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_a, M*K*sizeof(*a)));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_b, K*N*sizeof(*b)));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_c, M*N*sizeof(*c)));
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(M, K, sizeof(*a), a, M, dev_a, M), "hipblasSetMatrix A fail");
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(K, N, sizeof(*b), b, K, dev_b, K), "hipblasSetMatrix B fail");
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(M, N, sizeof(*c), c, M, dev_c, M), "hipblasSetMatrix C fail");

    hipDeviceSynchronize();
    start2 = clock();
    for (int i = 0; i < loops; ++i) {
        HANDLE_CUBLAS_ERROR(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, dev_a, M, dev_b, K, &beta, dev_c, M), "Sgemm fail");
    }
    hipDeviceSynchronize();
    stop2 = clock();
    HANDLE_CUBLAS_ERROR(hipblasGetMatrix(M, N, sizeof(*c), dev_c, M, c, M), "hipblasGetMatrix C fail");
    stop = clock();

    summarize_sgemm(c, loops, M, N, K, alpha, beta, start, stop, csv_output);
    if(!csv_output) {
        printf("ON DEVICE TIME:");
    }
    summarize_sgemm(c, loops, M, N, K, alpha, beta, start2, stop2, csv_output);
    if(csv_output) {
        printf("\n");
    }

    delete_float_matrix(a);
    delete_float_matrix(b);
    delete_float_matrix(c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipblasDestroy(handle);

    return 0;

}

int gpu_cublas_dgemm(int loops, int M, int N, int K, double alpha, double beta, bool csv_output)
{
    if(!csv_output) {
        printf("NVIDIA CUBLAS dgemm: loops=%d M=%d N=%d K=%d alpha=%f beta=%f\n", loops, M, N, K, alpha, beta);

        list_cuda_devices();
    } else {
        printf("NVIDIA CUBLAS dgemm,%d,%d,%d,%d,%f,%f",loops, M, N, K, alpha, beta);
    }

    hipblasHandle_t handle;
    HANDLE_CUBLAS_ERROR(hipblasCreate(&handle), "hipblasCreate fail");

    double *a, *b, *c;
    new_double_matrix(a, M, K);
    new_double_matrix(b, K, N);
    new_double_matrix(c, M, N);

    // time all the extra stuff for setting up the matrices
    clock_t start, stop;
    clock_t start2, stop2;
    start = clock();
    double *dev_a, *dev_b, *dev_c;
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_a, M*K*sizeof(*a)));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_b, K*N*sizeof(*b)));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_c, M*N*sizeof(*c)));
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(M, K, sizeof(*a), a, M, dev_a, M), "hipblasSetMatrix A fail");
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(K, N, sizeof(*b), b, K, dev_b, K), "hipblasSetMatrix B fail");
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(M, N, sizeof(*c), c, M, dev_c, M), "hipblasSetMatrix C fail");

    hipDeviceSynchronize();
    start2 = clock();
    for (int i = 0; i < loops; ++i) {
        HANDLE_CUBLAS_ERROR(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, dev_a, M, dev_b, K, &beta, dev_c, M), "Dgemm fail");
    }
    hipDeviceSynchronize();
    stop2 = clock();
    HANDLE_CUBLAS_ERROR(hipblasGetMatrix(M, N, sizeof(*c), dev_c, M, c, M), "hipblasGetMatrix C fail");
    stop = clock();

    summarize_dgemm(c, loops, M, N, K, alpha, beta, start, stop, csv_output);
    if(!csv_output) {
        printf("ON DEVICE TIME:");
    }
    summarize_dgemm(c, loops, M, N, K, alpha, beta, start2, stop2, csv_output);
    if(csv_output) {
        printf("\n");
    }

    delete_double_matrix(a);
    delete_double_matrix(b);
    delete_double_matrix(c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipblasDestroy(handle);

    return 0;

}

int gpu_cublasxt_sgemm(int loops, int M, int N, int K, float alpha, float beta, int block_dim, int num_gpus, int *gpu_ids, bool csv_output)
{
    if(!csv_output) {
        printf("NVIDIA CUBLASXT sgemm: loops=%d M=%d N=%d K=%d alpha=%f beta=%f block_dim=%d num_gpus=%d\n", loops, M, N, K, alpha, beta, block_dim, num_gpus);

        list_cuda_devices();
    } else {
        printf("NVIDIA CUBLASXT sgemm,%d,%d,%d,%d,%f,%f,%d,%d",loops, M, N, K, alpha, beta, block_dim, num_gpus);
    }

    cublasXtHandle_t handle;
    HANDLE_CUBLAS_ERROR(cublasXtCreate(&handle), "cublasXtCreate fail");

    HANDLE_CUBLAS_ERROR(cublasXtDeviceSelect(handle, num_gpus, gpu_ids), "cublasXtDeviceSelect fail");

    HANDLE_CUBLAS_ERROR(cublasXtSetBlockDim(handle, block_dim), "cublasXtSetBlockDim fail");

    float *a, *b, *c;
    new_float_matrix(a, M, K);
    new_float_matrix(b, K, N);
    new_float_matrix(c, M, N);

    clock_t start, stop;
    start = clock();
    for (int i = 0; i < loops; ++i) {
        HANDLE_CUBLAS_ERROR(cublasXtSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, a, M, b, K, &beta, c, M), "Sgemm fail");
    }
    stop = clock();

    summarize_sgemm(c, loops, M, N, K, alpha, beta, start, stop, csv_output);
    if(csv_output) {
        printf("\n");
    }

    delete_float_matrix(a);
    delete_float_matrix(b);
    delete_float_matrix(c);

    cublasXtDestroy(handle);

    return 0;

}

int gpu_cublasxt_dgemm(int loops, int M, int N, int K, double alpha, double beta, int block_dim, int num_gpus, int *gpu_ids, bool csv_output)
{
    if(!csv_output) {
        printf("NVIDIA CUBLASXT dgemm: loops=%d M=%d N=%d K=%d alpha=%f beta=%f block_dim=%d num_gpus=%d\n", loops, M, N, K, alpha, beta, block_dim, num_gpus);

        list_cuda_devices();
    } else {
        printf("NVIDIA CUBLASXT dgemm,%d,%d,%d,%d,%f,%f,%d,%d",loops, M, N, K, alpha, beta, block_dim, num_gpus);
    }

    cublasXtHandle_t handle;
    HANDLE_CUBLAS_ERROR(cublasXtCreate(&handle), "cublasXtCreate fail");

    HANDLE_CUBLAS_ERROR(cublasXtDeviceSelect(handle, num_gpus, gpu_ids), "cublasXtDeviceSelect fail");

    HANDLE_CUBLAS_ERROR(cublasXtSetBlockDim(handle, block_dim), "cublasXtSetBlockDim fail");

    double *a, *b, *c;
    new_double_matrix(a, M, K);
    new_double_matrix(b, K, N);
    new_double_matrix(c, M, N);

    clock_t start, stop;
    start = clock();
    for (int i = 0; i < loops; ++i) {
        HANDLE_CUBLAS_ERROR(cublasXtDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, a, M, b, K, &beta, c, M), "Dgemm fail");
    }
    stop = clock();

    summarize_dgemm(c, loops, M, N, K, alpha, beta, start, stop, csv_output);
    if(csv_output) {
        printf("\n");
    }

    delete_double_matrix(a);
    delete_double_matrix(b);
    delete_double_matrix(c);

    cublasXtDestroy(handle);

    return 0;

}

int gpu_cublas_ssyrkgemm(int loops, int M, int N, int K, float alpha, float beta, bool csv_output)
{
    if(!csv_output) {
        printf("NVIDIA CUBLAS ssyrkgemm: loops=%d M=%d N=%d K=%d alpha=%f beta=%f\n", loops, M, N, K, alpha, beta);

        list_cuda_devices();
    } else {
        printf("NVIDIA CUBLAS ssyrkgemm,%d,%d,%d,%d,%f,%f",loops, M, N, K, alpha, beta);
    }
    assert(M == N);

    hipblasHandle_t handle;
    HANDLE_CUBLAS_ERROR(hipblasCreate(&handle), "hipblasCreate fail");

    float *a, *b, *c;
    new_float_matrix(a, M, K);
    new_float_matrix(b, K, N);
    new_float_matrix(c, M, N);

    // time all the extra stuff for setting up the matrices
    clock_t start, stop;
    start = clock();
    float *dev_a, *dev_b, *dev_c;
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_a, M*K*sizeof(*a)));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_b, K*N*sizeof(*b)));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_c, M*N*sizeof(*c)));
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(M, K, sizeof(*a), a, M, dev_a, M), "hipblasSetMatrix A fail");
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(K, N, sizeof(*b), b, K, dev_b, K), "hipblasSetMatrix B fail");
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(M, N, sizeof(*c), c, M, dev_c, M), "hipblasSetMatrix C fail");

    for (int i = 0; i < loops; ++i) {
        HANDLE_CUBLAS_ERROR(hipblasSsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, N, K, &alpha, dev_a, M, &beta, dev_c, N), "Ssyrk fail");
        HANDLE_CUBLAS_ERROR(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, dev_a, M, dev_b, K, &beta, dev_c, M), "Sgemm fail");
    }
    HANDLE_CUBLAS_ERROR(hipblasGetMatrix(M, N, sizeof(*c), dev_c, M, c, M), "hipblasGetMatrix C fail");
    stop = clock();

    summarize_sgemm(c, loops, M, N, K, alpha, beta, start, stop, csv_output);
    if(csv_output) {
        printf("\n");
    }

    delete_float_matrix(a);
    delete_float_matrix(b);
    delete_float_matrix(c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipblasDestroy(handle);

    return 0;

}

int gpu_cublas_dsyrkgemm(int loops, int M, int N, int K, double alpha, double beta, bool csv_output)
{
    if(!csv_output) {
        printf("NVIDIA CUBLAS dsyrkgemm: loops=%d M=%d N=%d K=%d alpha=%f beta=%f\n", loops, M, N, K, alpha, beta);

        list_cuda_devices();
    } else {
        printf("NVIDIA CUBLAS dsyrkgemm,%d,%d,%d,%d,%f,%f",loops, M, N, K, alpha, beta);
    }
    assert(M == N);

    hipblasHandle_t handle;
    HANDLE_CUBLAS_ERROR(hipblasCreate(&handle), "hipblasCreate fail");

    double *a, *b, *c;
    new_double_matrix(a, M, K);
    new_double_matrix(b, K, N);
    new_double_matrix(c, M, N);

    // time all the extra stuff for setting up the matrices
    clock_t start, stop;
    start = clock();
    double *dev_a, *dev_b, *dev_c;
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_a, M*K*sizeof(*a)));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_b, K*N*sizeof(*b)));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_c, M*N*sizeof(*c)));
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(M, K, sizeof(*a), a, M, dev_a, M), "hipblasSetMatrix A fail");
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(K, N, sizeof(*b), b, K, dev_b, K), "hipblasSetMatrix B fail");
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(M, N, sizeof(*c), c, M, dev_c, M), "hipblasSetMatrix C fail");

    for (int i = 0; i < loops; ++i) {
		HANDLE_CUBLAS_ERROR(hipblasDsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, N, K, &alpha, dev_a, M, &beta, dev_c, N), "Dsyrk fail");
        HANDLE_CUBLAS_ERROR(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, dev_a, M, dev_b, K, &beta, dev_c, M), "Dgemm fail");
    }
    HANDLE_CUBLAS_ERROR(hipblasGetMatrix(M, N, sizeof(*c), dev_c, M, c, M), "hipblasGetMatrix C fail");
    stop = clock();

    summarize_dgemm(c, loops, M, N, K, alpha, beta, start, stop, csv_output);
    if(csv_output) {
        printf("\n");
    }

    delete_double_matrix(a);
    delete_double_matrix(b);
    delete_double_matrix(c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipblasDestroy(handle);

    return 0;

}

int gpu_cublasxt_ssyrkgemm(int loops, int M, int N, int K, float alpha, float beta, int block_dim, int num_gpus, int *gpu_ids, bool csv_output)
{
    if(!csv_output) {
        printf("NVIDIA CUBLASXT ssyrkgemm: loops=%d M=%d N=%d K=%d alpha=%f beta=%f block_dim=%d num_gpus=%d\n", loops, M, N, K, alpha, beta, block_dim, num_gpus);

        list_cuda_devices();
    } else {
         printf("NVIDIA CUBLASXT ssyrkgemm,%d,%d,%d,%d,%f,%f,%d,%d",loops, M, N, K, alpha, beta, block_dim, num_gpus);
    }
    assert(M == N);

    cublasXtHandle_t handle;
    HANDLE_CUBLAS_ERROR(cublasXtCreate(&handle), "cublasXtCreate fail");

    HANDLE_CUBLAS_ERROR(cublasXtDeviceSelect(handle, num_gpus, gpu_ids), "cublasXtDeviceSelect fail");

    HANDLE_CUBLAS_ERROR(cublasXtSetBlockDim(handle, block_dim), "cublasXtSetBlockDim fail");

    float *a, *b, *c;
    new_float_matrix(a, M, K);
    new_float_matrix(b, K, N);
    new_float_matrix(c, M, N);

    clock_t start, stop;
    start = clock();
    for (int i = 0; i < loops; ++i) {
        HANDLE_CUBLAS_ERROR(cublasXtSsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, N, K, &alpha, a, M, &beta, c, N), "Ssyrk fail");
        HANDLE_CUBLAS_ERROR(cublasXtSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, a, M, b, K, &beta, c, M), "Sgemm fail");
    }
    stop = clock();

    summarize_sgemm(c, loops, M, N, K, alpha, beta, start, stop, csv_output);
    if(csv_output) {
        printf("\n");
    }

    delete_float_matrix(a);
    delete_float_matrix(b);
    delete_float_matrix(c);

    cublasXtDestroy(handle);

    return 0;

}

int gpu_cublasxt_dsyrkgemm(int loops, int M, int N, int K, double alpha, double beta, int block_dim, int num_gpus, int *gpu_ids, bool csv_output)
{
    if(!csv_output) {
        printf("NVIDIA CUBLASXT dsyrkgemm: loops=%d M=%d N=%d K=%d alpha=%f beta=%f block_dim=%d num_gpus=%d\n", loops, M, N, K, alpha, beta, block_dim, num_gpus);

        list_cuda_devices();
    } else {
       printf("NVIDIA CUBLASXT dsyrkgemm,%d,%d,%d,%d,%f,%f,%d,%d",loops, M, N, K, alpha, beta, block_dim, num_gpus);
    }
    assert(M == N);

    cublasXtHandle_t handle;
    HANDLE_CUBLAS_ERROR(cublasXtCreate(&handle), "cublasXtCreate fail");

    HANDLE_CUBLAS_ERROR(cublasXtDeviceSelect(handle, num_gpus, gpu_ids), "cublasXtDeviceSelect fail");

    HANDLE_CUBLAS_ERROR(cublasXtSetBlockDim(handle, block_dim), "cublasXtSetBlockDim fail");

    double *a, *b, *c;
    new_double_matrix(a, M, K);
    new_double_matrix(b, K, N);
    new_double_matrix(c, M, N);

    clock_t start, stop;
    start = clock();
    for (int i = 0; i < loops; ++i) {
        HANDLE_CUBLAS_ERROR(cublasXtDsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, N, K, &alpha, a, M, &beta, c, N), "Dsyrk fail");
        HANDLE_CUBLAS_ERROR(cublasXtDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, a, M, b, K, &beta, c, M), "Dgemm fail");
    }
    stop = clock();

    summarize_dgemm(c, loops, M, N, K, alpha, beta, start, stop, csv_output);
    if(csv_output) {
        printf("\n");
    }

    delete_double_matrix(a);
    delete_double_matrix(b);
    delete_double_matrix(c);

    cublasXtDestroy(handle);

    return 0;

}
