#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <stdio.h>
#include <windows.h>
#include <hipblas.h>
#include "gpu_blas_test.h"
#include "util.h"

#define HANDLE_CUDA_ERROR( err ) ( HandleCudaError( err, __FILE__, __LINE__ ) )
static void HandleCudaError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_CUBLAS_ERROR( err, str ) ( HandleCublasError( err, __FILE__, __LINE__, str) )
static void HandleCublasError(hipblasStatus_t err, const char *file, int line, const char *str)
{
    if (err != HIPBLAS_STATUS_SUCCESS)
    {
        printf("error %s %d in %s at line %d\n", str, err, // FIXME why no error code?
            file, line);
        exit(EXIT_FAILURE);
    }
}

int main_gpu_test(int loops, int M, int N, int K)
{
    printf("CUBLAS sgemm: loops=%d M=%d N=%d K=%d\n", loops, M, N, K);
    
    hipblasHandle_t handle;
    HANDLE_CUBLAS_ERROR(hipblasCreate(&handle),"hipblasCreate fail");

    float *a, *b, *c;
    new_float_matrix(a, M, K);
    new_float_matrix(b, K, N);
    new_float_matrix(c, M, N);

    // time all the extra stuff for setting up the matrices
    clock_t start, stop;
    start = clock();
    float *dev_a, *dev_b, *dev_c;
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_a, M*K*sizeof(*a)));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_b, K*N*sizeof(*b)));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&dev_c, M*N*sizeof(*c)));
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(M, K, sizeof(*a), a, M, dev_a, M), "hipblasSetMatrix A fail");
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(K, N, sizeof(*b), b, K, dev_b, K), "hipblasSetMatrix B fail");
    HANDLE_CUBLAS_ERROR(hipblasSetMatrix(M, N, sizeof(*c), c, M, dev_c, M), "hipblasSetMatrix C fail");

    float alpha = 1.11f, beta = 0.91f;
    for (int i = 0; i < loops; ++i) {
        // FIXME this is column-major, CPU is row-major
        HANDLE_CUBLAS_ERROR(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, dev_a, M, dev_b, K, &beta, dev_c, M), "Sgemm fail");
    }
    HANDLE_CUBLAS_ERROR(hipblasGetMatrix(M, N, sizeof(*c), dev_c, M, c, M), "hipblasGetMatrix C fail");
    stop = clock();

    printf("sgemm_multiply(). Elapsed time = %g seconds\n",
        ((double)(stop - start)) / CLOCKS_PER_SEC);

    printf("C:\n");
    pr_array(c, N);

    delete_float_matrix(a);
    delete_float_matrix(b);
    delete_float_matrix(c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipblasDestroy(handle);

    return 0;

}
